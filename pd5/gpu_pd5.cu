#include "hip/hip_runtime.h"
/*
 * cuda_pd5.cpp
 *
 *  Contains GPU kernel code for PD5.
 *
 *  Derived from the RSA Data Security, Inc. MD5 Message Digest Algorithm
 *
 *  Some code also used with permission of Mario Juric: http://majuric.org/
 *
 */



#include <stdio.h>
#include <sys/time.h>

#include "cuda_pd5.hpp"

typedef unsigned int uint;

//
// On-device variable declarations
//

extern __shared__ uint memory[];	    // on-chip shared memory
__constant__ uint k[64], rconst[16];	// constants (in fast on-chip constant cache)

//
// MD5 magic numbers. These will be loaded into on-device "constant" memory
//
static const uint k_cpu[64] =
{
	0xd76aa478, 	0xe8c7b756,	0x242070db,	0xc1bdceee,
	0xf57c0faf,	0x4787c62a, 	0xa8304613,	0xfd469501,
	0x698098d8,	0x8b44f7af,	0xffff5bb1,	0x895cd7be,
	0x6b901122, 	0xfd987193, 	0xa679438e,	0x49b40821,

	0xf61e2562,	0xc040b340, 	0x265e5a51, 	0xe9b6c7aa,
	0xd62f105d,	0x2441453,	0xd8a1e681,	0xe7d3fbc8,
	0x21e1cde6,	0xc33707d6, 	0xf4d50d87, 	0x455a14ed,
	0xa9e3e905,	0xfcefa3f8, 	0x676f02d9, 	0x8d2a4c8a,

	0xfffa3942,	0x8771f681, 	0x6d9d6122, 	0xfde5380c,
	0xa4beea44, 	0x4bdecfa9, 	0xf6bb4b60, 	0xbebfbc70,
	0x289b7ec6, 	0xeaa127fa, 	0xd4ef3085,	0x4881d05,
	0xd9d4d039, 	0xe6db99e5, 	0x1fa27cf8, 	0xc4ac5665,

	0xf4292244, 	0x432aff97, 	0xab9423a7, 	0xfc93a039,
	0x655b59c3, 	0x8f0ccc92, 	0xffeff47d, 	0x85845dd1,
	0x6fa87e4f, 	0xfe2ce6e0, 	0xa3014314, 	0x4e0811a1,
	0xf7537e82, 	0xbd3af235, 	0x2ad7d2bb, 	0xeb86d391,
};

static const uint rconst_cpu[16] =
{
	7, 12, 17, 22,   5,  9, 14, 20,   4, 11, 16, 23,   6, 10, 15, 21
};

void init_constants()
{
	hipMemcpyToSymbol(HIP_SYMBOL(k), k_cpu, sizeof(k));
	hipMemcpyToSymbol(HIP_SYMBOL(rconst), rconst_cpu, sizeof(rconst));
}

//
// MD5 routines (straight from Wikipedia's MD5 pseudocode description)
//

__device__ inline uint leftrotate (uint x, uint c)
{
	return (x << c) | (x >> (32-c));
}

__device__ inline uint r(const uint i)
{
	return rconst[(i / 16) * 4 + i % 4];
}

// Accessor for w[16] array. Naively, this would just be w[i]; however, this
// choice leads to worst-case-scenario access pattern wrt. shared memory
// bank conflicts, as the same indices in different threads fall into the
// same bank (as the words are 16 uints long). The packing below causes the
// same indices in different threads of a warp to map to different banks. In
// testing this gave a ~40% speedup.
//
// PS: An alternative solution would be to make the w array 17 uints long
// (thus wasting a little shared memory)
//
__device__ inline uint &getw(uint *w, const int i)
{
	return w[(i+threadIdx.x) % 16];
}

__device__ inline uint getw(const uint *w, const int i)	// const- version
{
	return w[(i+threadIdx.x) % 16];
}


__device__ inline uint getk(const int i)
{
	return k[i];	// Note: this is as fast as possible (measured)
}

__device__ void step(const uint i, const uint f, const uint g, uint &a, uint &b, uint &c, uint &d, const uint *w)
{
	uint temp = d;
	d = c;
	c = b;
	b = b + leftrotate((a + f + getk(i) + getw(w, g)), r(i));
	a = temp;
}



//////////////////////////////////////////////////////////////////////////////
/////////////       Ron Rivest's MD5 C Implementation       //////////////////
//////////////////////////////////////////////////////////////////////////////

/*
 **********************************************************************
 ** Copyright (C) 1990, RSA Data Security, Inc. All rights reserved. **
 **                                                                  **
 ** License to copy and use this software is granted provided that   **
 ** it is identified as the "RSA Data Security, Inc. MD5 Message     **
 ** Digest Algorithm" in all material mentioning or referencing this **
 ** software or this function.                                       **
 **                                                                  **
 ** License is also granted to make and use derivative works         **
 ** provided that such works are identified as "derived from the RSA **
 ** Data Security, Inc. MD5 Message Digest Algorithm" in all         **
 ** material mentioning or referencing the derived work.             **
 **                                                                  **
 ** RSA Data Security, Inc. makes no representations concerning      **
 ** either the merchantability of this software or the suitability   **
 ** of this software for any particular purpose.  It is provided "as **
 ** is" without express or implied warranty of any kind.             **
 **                                                                  **
 ** These notices must be retained in any copies of any part of this **
 ** documentation and/or software.                                   **
 **********************************************************************
 */


/* F, G and H are basic MD5 functions: selection, majority, parity */
#define F(x, y, z) (((x) & (y)) | ((~x) & (z)))
#define G(x, y, z) (((x) & (z)) | ((y) & (~z)))
#define H(x, y, z) ((x) ^ (y) ^ (z))
#define I(x, y, z) ((y) ^ ((x) | (~z))) 

/* ROTATE_LEFT rotates x left n bits */
#define ROTATE_LEFT(x, n) (((x) << (n)) | ((x) >> (32-(n))))

/* FF, GG, HH, and II transformations for rounds 1, 2, 3, and 4 */
/* Rotation is separate from addition to prevent recomputation */
#define FF(a, b, c, d, x, s, ac) \
  {(a) += F ((b), (c), (d)) + (x) + (uint)(ac); \
   (a) = ROTATE_LEFT ((a), (s)); \
   (a) += (b); \
  }
#define GG(a, b, c, d, x, s, ac) \
  {(a) += G ((b), (c), (d)) + (x) + (uint)(ac); \
   (a) = ROTATE_LEFT ((a), (s)); \
   (a) += (b); \
  }
#define HH(a, b, c, d, x, s, ac) \
  {(a) += H ((b), (c), (d)) + (x) + (uint)(ac); \
   (a) = ROTATE_LEFT ((a), (s)); \
   (a) += (b); \
  }
#define II(a, b, c, d, x, s, ac) \
  {(a) += I ((b), (c), (d)) + (x) + (uint)(ac); \
   (a) = ROTATE_LEFT ((a), (s)); \
   (a) += (b); \
  }


/* Basic MD5 step. Transform buf based on in.
 */
void inline __device__ md5_v2(const uint *in, uint &a, uint &b, uint &c, uint &d)
{
	//Initialize hash value for this chunk:
	uint a0 = a;
	uint b0 = b;
	uint c0 = c;
	uint d0 = d;


  /* Round 1 */
#define S11 7
#define S12 12
#define S13 17
#define S14 22
  FF ( a, b, c, d, getw(in,  0), S11, 3614090360); /* 1 */
  FF ( d, a, b, c, getw(in,  1), S12, 3905402710); /* 2 */
  FF ( c, d, a, b, getw(in,  2), S13,  606105819); /* 3 */
  FF ( b, c, d, a, getw(in,  3), S14, 3250441966); /* 4 */
  FF ( a, b, c, d, getw(in,  4), S11, 4118548399); /* 5 */
  FF ( d, a, b, c, getw(in,  5), S12, 1200080426); /* 6 */
  FF ( c, d, a, b, getw(in,  6), S13, 2821735955); /* 7 */
  FF ( b, c, d, a, getw(in,  7), S14, 4249261313); /* 8 */
  FF ( a, b, c, d, getw(in,  8), S11, 1770035416); /* 9 */
  FF ( d, a, b, c, getw(in,  9), S12, 2336552879); /* 10 */
  FF ( c, d, a, b, getw(in, 10), S13, 4294925233); /* 11 */
  FF ( b, c, d, a, getw(in, 11), S14, 2304563134); /* 12 */
  FF ( a, b, c, d, getw(in, 12), S11, 1804603682); /* 13 */
  FF ( d, a, b, c, getw(in, 13), S12, 4254626195); /* 14 */
  FF ( c, d, a, b, getw(in, 14), S13, 2792965006); /* 15 */
  FF ( b, c, d, a, getw(in, 15), S14, 1236535329); /* 16 */
 
  /* Round 2 */
#define S21 5
#define S22 9
#define S23 14
#define S24 20
  GG ( a, b, c, d, getw(in,  1), S21, 4129170786); /* 17 */
  GG ( d, a, b, c, getw(in,  6), S22, 3225465664); /* 18 */
  GG ( c, d, a, b, getw(in, 11), S23,  643717713); /* 19 */
  GG ( b, c, d, a, getw(in,  0), S24, 3921069994); /* 20 */
  GG ( a, b, c, d, getw(in,  5), S21, 3593408605); /* 21 */
  GG ( d, a, b, c, getw(in, 10), S22,   38016083); /* 22 */
  GG ( c, d, a, b, getw(in, 15), S23, 3634488961); /* 23 */
  GG ( b, c, d, a, getw(in,  4), S24, 3889429448); /* 24 */
  GG ( a, b, c, d, getw(in,  9), S21,  568446438); /* 25 */
  GG ( d, a, b, c, getw(in, 14), S22, 3275163606); /* 26 */
  GG ( c, d, a, b, getw(in,  3), S23, 4107603335); /* 27 */
  GG ( b, c, d, a, getw(in,  8), S24, 1163531501); /* 28 */
  GG ( a, b, c, d, getw(in, 13), S21, 2850285829); /* 29 */
  GG ( d, a, b, c, getw(in,  2), S22, 4243563512); /* 30 */
  GG ( c, d, a, b, getw(in,  7), S23, 1735328473); /* 31 */
  GG ( b, c, d, a, getw(in, 12), S24, 2368359562); /* 32 */

  /* Round 3 */
#define S31 4
#define S32 11
#define S33 16
#define S34 23
  HH ( a, b, c, d, getw(in,  5), S31, 4294588738); /* 33 */
  HH ( d, a, b, c, getw(in,  8), S32, 2272392833); /* 34 */
  HH ( c, d, a, b, getw(in, 11), S33, 1839030562); /* 35 */
  HH ( b, c, d, a, getw(in, 14), S34, 4259657740); /* 36 */
  HH ( a, b, c, d, getw(in,  1), S31, 2763975236); /* 37 */
  HH ( d, a, b, c, getw(in,  4), S32, 1272893353); /* 38 */
  HH ( c, d, a, b, getw(in,  7), S33, 4139469664); /* 39 */
  HH ( b, c, d, a, getw(in, 10), S34, 3200236656); /* 40 */
  HH ( a, b, c, d, getw(in, 13), S31,  681279174); /* 41 */
  HH ( d, a, b, c, getw(in,  0), S32, 3936430074); /* 42 */
  HH ( c, d, a, b, getw(in,  3), S33, 3572445317); /* 43 */
  HH ( b, c, d, a, getw(in,  6), S34,   76029189); /* 44 */
  HH ( a, b, c, d, getw(in,  9), S31, 3654602809); /* 45 */
  HH ( d, a, b, c, getw(in, 12), S32, 3873151461); /* 46 */
  HH ( c, d, a, b, getw(in, 15), S33,  530742520); /* 47 */
  HH ( b, c, d, a, getw(in,  2), S34, 3299628645); /* 48 */

  /* Round 4 */
#define S41 6
#define S42 10
#define S43 15
#define S44 21
  II ( a, b, c, d, getw(in,  0), S41, 4096336452); /* 49 */
  II ( d, a, b, c, getw(in,  7), S42, 1126891415); /* 50 */
  II ( c, d, a, b, getw(in, 14), S43, 2878612391); /* 51 */
  II ( b, c, d, a, getw(in,  5), S44, 4237533241); /* 52 */
  II ( a, b, c, d, getw(in, 12), S41, 1700485571); /* 53 */
  II ( d, a, b, c, getw(in,  3), S42, 2399980690); /* 54 */
  II ( c, d, a, b, getw(in, 10), S43, 4293915773); /* 55 */
  II ( b, c, d, a, getw(in,  1), S44, 2240044497); /* 56 */
  II ( a, b, c, d, getw(in,  8), S41, 1873313359); /* 57 */
  II ( d, a, b, c, getw(in, 15), S42, 4264355552); /* 58 */
  II ( c, d, a, b, getw(in,  6), S43, 2734768916); /* 59 */
  II ( b, c, d, a, getw(in, 13), S44, 1309151649); /* 60 */
  II ( a, b, c, d, getw(in,  4), S41, 4149444226); /* 61 */
  II ( d, a, b, c, getw(in, 11), S42, 3174756917); /* 62 */
  II ( c, d, a, b, getw(in,  2), S43,  718787259); /* 63 */
  II ( b, c, d, a, getw(in,  9), S44, 3951481745); /* 64 */

	a += a0;
	b += b0;
	c += c0;
	d += d0;

}

//////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////

/* Function calculates PD5, very similar to MD5, however it is not
 * done sequentially.  The Initial values have the thread id (same
 * as block number) added on to the initial seed values.
 */
__global__ void pd5_calc(uint *gwords, uint *hash, int realthreads, int lThreadOffset )
{
	unsigned int a, b, c, d;
	
	int linidx = (gridDim.x*blockIdx.y + blockIdx.x)*blockDim.x + threadIdx.x; // assuming blockDim.y = 1 and threadIdx.y = 0, always
	if(linidx >= realthreads) { return; } // this check slows down the code by ~0.4% (measured)

	// load the dictionary word for this thread
	uint *word = &memory[0] + threadIdx.x*16;
	for(int i=0; i != 16; i++)
	{
		getw(word, i) = gwords[(linidx)*16+i];
	}

	// compute MD5 hash
    a = 0x67452301 + linidx + lThreadOffset;
    b = 0xEFCDAB89 + linidx + lThreadOffset;
    c = 0x98BADCFE + linidx + lThreadOffset;
    d = 0x10325476 + linidx + lThreadOffset;

	md5_v2(word, a, b, c, d);

	// return the hash
	linidx *= 4;
	hash[(linidx)+0] = a;
	hash[(linidx)+1] = b;
	hash[(linidx)+2] = c;
	hash[(linidx)+3] = d;
}


/* Function to collapse all the block results down to one hash value.
 */
__global__ void pd5_add(uint *hash, int realthreads, int lOffset )
{
    int linidx = (gridDim.x*blockIdx.y + blockIdx.x)*blockDim.x + threadIdx.x; // assuming blockDim.y = 1 and threadIdx.y = 0, always
    if(linidx >= realthreads) { return; } // this check slows down the code by ~0.4% (measured)

    linidx *= 4;
    hash[(linidx)+0] += hash[(linidx)+lOffset];
    hash[(linidx)+1] += hash[(linidx)+lOffset+1];
    hash[(linidx)+2] += hash[(linidx)+lOffset+2];
    hash[(linidx)+3] += hash[(linidx)+lOffset+3];
}

// A helper to export the kernel call to C++ code not compiled with nvcc
double execute_kernel(int blocks_x, int blocks_y, int threads_per_block, int shared_mem_required, int realthreads, int lThreadOffset, uint *gpuWords, uint *gpuHashes )
{
	struct timeval Start;
	struct timeval End;
	double dRet = 0.0;
	
	dim3 grid;
	grid.x = blocks_x; grid.y = blocks_y;

	gettimeofday( &Start, NULL );


    hipDeviceSynchronize();

    /* use a different block every time you call the function */
    pd5_calc<<<grid, threads_per_block, shared_mem_required>>>(gpuWords, gpuHashes, realthreads, lThreadOffset );

    hipDeviceSynchronize();

	gettimeofday( &End, NULL );

	dRet = (double)(End.tv_sec - Start.tv_sec) * 1000.0;
	dRet += (double)(End.tv_usec - Start.tv_usec) / 1000.0;

	//printf("Int Kernel time: %lf, \n", dRet );


    int lHalf = 1;

    /* Find the power of two closest to half the real threads */
    while( lHalf < realthreads )
        lHalf <<= 1;
    lHalf >>= 1; /* Move back on bit */

    //printf( "lHalf: %i, realthreads %i, %i\n", lHalf, realthreads, realthreads-lHalf );

    /* Check if we need to recombine results in the GPU */
    if( realthreads <= USE_CPU )
        return dRet;

    /* Combine results, first step will bring us down to a power of 2, (offset is for 4byte hahses) */
    pd5_add<<<grid, threads_per_block, shared_mem_required>>>(gpuHashes, realthreads-lHalf, lHalf*4 );
    hipDeviceSynchronize();

    realthreads = lHalf;

    /* Cut the results in half until it is more efficient to use the CPU */
    while( realthreads > USE_CPU )
    {
        /* Only use half the threads to add the results */
        realthreads >>= 1;
        /* Combine results */
        pd5_add<<<grid, threads_per_block, shared_mem_required>>>(gpuHashes, realthreads, realthreads*4 );
        hipDeviceSynchronize();
    }

	return dRet;
}

